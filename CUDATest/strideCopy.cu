#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define SIZE 128*1024*1024
#define BLOCK_SIZE 1024

__global__ void offsetCopy(float *odata, const float *idata, int offset)
{
	int xid = blockIdx.x * blockDim.x + threadIdx.x + offset;
	odata[xid] = idata[xid];
}

__global__ void strideCopy(float *odata, const float *idata, int stride)
{
	int xid = (blockIdx.x * blockDim.x + threadIdx.x) * stride;
	odata[xid] = idata[xid];
}

int main()
{
	srand((unsigned)time(NULL));
	
	int i;
	float* cpu_A = (float*)malloc(sizeof(float)*SIZE);
	float* cpu_B = (float*)malloc(sizeof(float)*SIZE);
	memset(cpu_B, 0, sizeof(float)*SIZE);
	for (i = 0; i < SIZE; ++i){
		cpu_A[i] = (float)(rand() / RAND_MAX);
	}

	float *dev_a;
	float *dev_b;

	hipSetDevice(0);

	hipMalloc((void**)&dev_a, SIZE * sizeof(float) * 10);
	hipMalloc((void**)&dev_b, SIZE * sizeof(float) * 10);
	hipMemcpy(dev_a, cpu_A, SIZE * sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (i = 1; i <= 10; ++i)
	{
		hipEventRecord(start, 0);
		
		int blocks = SIZE / BLOCK_SIZE;
		int threads = BLOCK_SIZE;
		strideCopy<<<blocks, threads>>>(dev_b, dev_a, i);

		hipDeviceSynchronize();

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTime, start, stop);
		printf("GPU use time: %f (ms), Stride: %d \n", elapsedTime, i);
	}

	hipFree(dev_a);
	hipFree(dev_b);
}
