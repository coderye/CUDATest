#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BLOCK_SIZE	32
#define WA		16384
#define HA		4096
#define WB		2048
#define HB		WA
#define WC		WB
#define HC		HA

void matrixMulGPU(float* _C, const float *_A, const float *_B, int _wa, int _ha, int _wb);
void randomInit(float* _data, int _size);

__global__ void matrix_kernel(float* _C, const float* _A, const float *_B, int _wa, int _wb)
{
	float sum = 0;
	//找出该线程所在的行列
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	//线程Thread(row,col)负责计算C(row,col)
	for (int i = 0; i < _wa; ++i)
	{
		sum += _A[row*_wa + i] * _B[i*_wb + col];
	}
	_C[row*_wb + col] = sum;
}

int main()
{
	srand((unsigned)time(NULL));

	float* cpu_A = (float*)malloc(sizeof(float)*WA*HA);
	float* cpu_B = (float*)malloc(sizeof(float)*WB*HB);

	randomInit(cpu_A, WA*HA);
	randomInit(cpu_B, WB*HB);

	float* cpu_C = (float*)malloc(sizeof(float)*WC*HC);
	memset(cpu_C, 0, sizeof(float)*WC*HC);

	//----- Matrix Mul w/ GPU
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	matrixMulGPU(cpu_C, cpu_A, cpu_B, WA, HA, WB);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("GPU use time: %f (ms), Block Size: %d \n", elapsedTime, BLOCK_SIZE);
	//-----

	hipEventDestroy(start);
	hipEventDestroy(stop);

	free(cpu_A);
	free(cpu_B);
	free(cpu_C);

	hipDeviceReset();
	return 0;
}

void matrixMulGPU(float* _C, const float *_A, const float *_B, int _wa, int _ha, int _wb)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;

	hipSetDevice(0);

	hipMalloc((void**)&dev_a, _wa * _ha * sizeof(float));
	hipMalloc((void**)&dev_b, _wb * _wa * sizeof(float));
	hipMalloc((void**)&dev_c, _wb * _ha * sizeof(float));

	hipMemcpy(dev_a, _A, _wa * _ha * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, _B, _wb * _wa * sizeof(float), hipMemcpyHostToDevice);

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(WC / BLOCK_SIZE, HC / BLOCK_SIZE);
	matrix_kernel << <blocks, threads >> >(dev_c, dev_a, dev_b, _wa, _wb);

	hipDeviceSynchronize();
	hipMemcpy(_C, dev_c, _wb * _ha * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

void randomInit(float* _data, int _size)
{
	for (int i = 0; i < _size; ++i)
	{
		_data[i] = rand() / (float)RAND_MAX;
	}
}