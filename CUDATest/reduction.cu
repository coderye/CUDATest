#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BLOCK_SIZE	32
#define WA		16384
#define HA		4096
#define WB		2048
#define HB		WA
#define WC		WB
#define HC		HA

void matrixMulGPU(float* _C, const float *_A, const float *_B, int _wa, int _ha, int _wb);
void randomInit(float* _data, int _size);

__global__ void reduction_kernel(float* _odata, const float* _idata, const unsigned int _size)
{
	__shared__ float partialSum[];
	unsigned int t = threadIdx.x;
	//unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

	//partialSum[t] = (i < _size) ? _idata[i] : 0;
	float sum = i < _size) ? _idata[i] : 0;
	if (i + blockDim.x < _size)
		sum += _idata[i + blockDim.x];
	partialSum[t] = sum;
	
	unsigned int stride;
	for (stride = blockDim.x/2; stride > 0; stride >>= 1)
	{
		__syncthreads();
		if (t < stride)
			//partialSum[t] += partialSum[t + stride];
			partialSum[t] = sum = sum + partialSum[t + stride];
	}
	
	if (t == 0)
		_odata[blockIdx.x] = partialSum[0];
}

int main()
{
	srand((unsigned)time(NULL));

	float* cpu_A = (float*)malloc(sizeof(float)*WA*HA);
	float* cpu_B = (float*)malloc(sizeof(float)*WB*HB);

	randomInit(cpu_A, WA*HA);
	randomInit(cpu_B, WB*HB);

	float* cpu_C = (float*)malloc(sizeof(float)*WC*HC);
	memset(cpu_C, 0, sizeof(float)*WC*HC);

	//----- Matrix Mul w/ GPU
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	matrixMulGPU(cpu_C, cpu_A, cpu_B, WA, HA, WB);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("GPU use time: %f (ms), Block Size: %d \n", elapsedTime, BLOCK_SIZE);
	//-----

	hipEventDestroy(start);
	hipEventDestroy(stop);

	free(cpu_A);
	free(cpu_B);
	free(cpu_C);

	hipDeviceReset();
	return 0;
}

void matrixMulGPU(float* _C, const float *_A, const float *_B, int _wa, int _ha, int _wb)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;

	hipSetDevice(0);

	hipMalloc((void**)&dev_a, _wa * _ha * sizeof(float));
	hipMalloc((void**)&dev_b, _wb * _wa * sizeof(float));
	hipMalloc((void**)&dev_c, _wb * _ha * sizeof(float));

	hipMemcpy(dev_a, _A, _wa * _ha * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, _B, _wb * _wa * sizeof(float), hipMemcpyHostToDevice);

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(WC / BLOCK_SIZE, HC / BLOCK_SIZE);
	matrix_kernel << <blocks, threads >> >(dev_c, dev_a, dev_b, _wa, _wb);

	hipDeviceSynchronize();
	hipMemcpy(_C, dev_c, _wb * _ha * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

void randomInit(float* _data, int _size)
{
	for (int i = 0; i < _size; ++i)
	{
		_data[i] = rand() / (float)RAND_MAX;
	}
}