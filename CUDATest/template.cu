#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BLOCK_SIZE	32
#define BLOCK_SIZE2	32//64//16
#define WA		16384
#define HA		4096
#define WB		2048
#define HB		WA
#define WC		WB
#define HC		HA

void matrixMulGPU(float* _C, const float *_A, const float *_B, int _wa, int _ha, int _wb);
void matrixMulGPU2(float* _C, const float *_A, const float *_B, int _wa, int _ha, int _wb);
void randomInit(float* _data, int _size);
bool checkError(const float* _A, const float* _B, int _size);

__global__ void matrix_kernel(float* _C, const float* _A, const float *_B, int _wa, int _wb)
{
	float sum = 0;
	//找出该线程所在的行列
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	//线程Thread(row,col)负责计算C(row,col)
	for (int i = 0; i < _wa; ++i)
	{
		sum += _A[row*_wa + i] * _B[i*_wb + col];
	}
	_C[row*_wb + col] = sum;
}

__global__ void matrix_kernel_shared(float* _C, const float* _A, const float *_B, int _wa, int _wb)
{
	__shared__ float As[BLOCK_SIZE2][BLOCK_SIZE2];
	__shared__ float Bs[BLOCK_SIZE2][BLOCK_SIZE2];

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//该block要处理的A
	int aBegin = _wa*(by*BLOCK_SIZE2);//A(0,by)
	int aEnd = aBegin + _wa - 1;
	int aStep = BLOCK_SIZE2;//offsetA

	int bBegin = BLOCK_SIZE2*bx;//B(bx,0)
	int bStep = BLOCK_SIZE2*_wb;//offsetB

	float cSub = 0;
	for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
	{
		//每个线程负责一个元素拷贝
		As[ty][tx] = _A[a + _wa*ty + tx];
		Bs[ty][tx] = _B[b + _wb*ty + tx];

		__syncthreads();

		//每个线程负责计算一个子块i 和 子块j的子乘积
		for (int k = 0; k < BLOCK_SIZE2; ++k)
		{
			cSub += As[ty][k] * Bs[k][tx];
		}

		__syncthreads();
	}

	//全局地址，向全局寄存器写回去
	//一个线程负责一个元素，一个block负责一个子块
	int cIndex = (by*BLOCK_SIZE2 + ty)*_wb + (bx*BLOCK_SIZE2 + tx);
	_C[cIndex] = cSub;
}


int main()
{
	srand((unsigned)time(NULL));

	float* cpu_A = (float*)malloc(sizeof(float)*WA*HA);
	float* cpu_B = (float*)malloc(sizeof(float)*WB*HB);

	randomInit(cpu_A, WA*HA);
	randomInit(cpu_B, WB*HB);

	float* cpu_C = (float*)malloc(sizeof(float)*WC*HC);
	memset(cpu_C, 0, sizeof(float)*WC*HC);

	float* cpu_C2 = (float*)malloc(sizeof(float)*WC*HC);
	memset(cpu_C2, 0, sizeof(float)*WC*HC);

	//----- Matrix Mul w/ GPU
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	matrixMulGPU(cpu_C, cpu_A, cpu_B, WA, HA, WB);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("GPU use time: %f (ms), Block Size: %d \n", elapsedTime, BLOCK_SIZE);
	//-----

	//----- Matrix Mul w/ GPU Shared Memory
	hipEventRecord(start, 0);

	matrixMulGPU2(cpu_C, cpu_A, cpu_B, WA, HA, WB);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("GPU use time: %f (ms), w/ shared memory, Block Size: %d\n", elapsedTime, BLOCK_SIZE2);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	free(cpu_A);
	free(cpu_B);
	free(cpu_C);
	free(cpu_C2);

	hipDeviceReset();
	return 0;
}

void matrixMulGPU(float* _C, const float *_A, const float *_B, int _wa, int _ha, int _wb)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;

	hipSetDevice(0);

	hipMalloc((void**)&dev_a, _wa * _ha * sizeof(float));
	hipMalloc((void**)&dev_b, _wb * _wa * sizeof(float));
	hipMalloc((void**)&dev_c, _wb * _ha * sizeof(float));

	hipMemcpy(dev_a, _A, _wa * _ha * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, _B, _wb * _wa * sizeof(float), hipMemcpyHostToDevice);

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(WC / BLOCK_SIZE, HC / BLOCK_SIZE);
	matrix_kernel << <blocks, threads >> >(dev_c, dev_a, dev_b, _wa, _wb);

	hipDeviceSynchronize();
	hipMemcpy(_C, dev_c, _wb * _ha * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

void matrixMulGPU2(float* _C, const float *_A, const float *_B, int _wa, int _ha, int _wb)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;

	hipSetDevice(0);

	hipMalloc((void**)&dev_a, _wa * _ha * sizeof(float));
	hipMalloc((void**)&dev_b, _wb * _wa * sizeof(float));
	hipMalloc((void**)&dev_c, _wb * _ha * sizeof(float));

	hipMemcpy(dev_a, _A, _wa * _ha * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, _B, _wb * _wa * sizeof(float), hipMemcpyHostToDevice);

	dim3 threads(BLOCK_SIZE2, BLOCK_SIZE2);
	dim3 blocks(WC / (BLOCK_SIZE2), HC / (BLOCK_SIZE2));
	matrix_kernel_shared << <blocks, threads >> >(dev_c, dev_a, dev_b, _wa, _wb);

	hipDeviceSynchronize();
	hipMemcpy(_C, dev_c, _wb * _ha * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

void randomInit(float* _data, int _size)
{
	for (int i = 0; i < _size; ++i)
	{
		_data[i] = rand() / (float)RAND_MAX;
	}
}

bool checkError(const float* _A, const float* _B, int _size)
{
	for (int i = 0; i < _size; ++i)
	{
		if (fabs(_A[i] - _B[i]) > 1.0e-3)
		{
			printf("%f \t %f\n", _A[i], _B[i]);
			return false;
		}
	}
	return true;
}
