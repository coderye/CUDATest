#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define SIZE 128*1024*1024
#define BLOCK_SIZE 1024

__global__ void offsetCopy(float *odata, const float *idata, int offset)
{
	int xid = blockIdx.x * blockDim.x + threadIdx.x + offset;
	for (int i = 0; i < 2; i++)
		odata[xid+i] = idata[xid+i];
}

__global__ void strideCopy(float *odata, const float *idata, int stride)
{
	int xid = (blockIdx.x * blockDim.x + threadIdx.x) * stride;
	odata[xid] = idata[xid];
}

int main()
{
	srand((unsigned)time(NULL));
	
	int i;
	float* cpu_A = (float*)malloc(sizeof(float)*SIZE);
	float* cpu_B = (float*)malloc(sizeof(float)*SIZE);
	memset(cpu_B, 0, sizeof(float)*SIZE);
	for (i = 0; i < SIZE; ++i){
		cpu_A[i] = (float)(rand() / RAND_MAX);
	}

	float *dev_a;
	float *dev_b;

	hipSetDevice(0);

	hipMalloc((void**)&dev_a, SIZE * sizeof(float) * 2);
	hipMalloc((void**)&dev_b, SIZE * sizeof(float) * 2);
	hipMemcpy(dev_a, cpu_A, SIZE * sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (i = 1; i <= 10240; i *= 10)
	{
		hipEventRecord(start, 0);
		
		int blocks = SIZE / BLOCK_SIZE;
		int threads = BLOCK_SIZE;
		offsetCopy<<<blocks, threads>>>(dev_b, dev_a, i-1);
		//strideCopy << <SIZE / BLOCK_SIZE, BLOCK_SIZE >> >(dev_b, dev_a, 0);

		hipDeviceSynchronize();

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTime, start, stop);
		printf("GPU use time: %f (ms), Step: %d \n", elapsedTime, i-1);
	}

	hipFree(dev_a);
	hipFree(dev_b);
}
